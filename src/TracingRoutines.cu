#include "TracingRoutines.h"

#include <glm/glm.hpp>
#include <hip/hip_runtime.h>
#include "Ray.h"
#include "HitInfo.h"
#include "Material.h"
#include "Scene.h"

__host__ __device__
HitInfo hitScene(const Ray& ray, RenderParams& params)
{
    HitInfo closestHit;
    for (size_t i = 0; i < params.spheresCount; i++)
    {
        Sphere* sphere = params.spheres + i;
        HitInfo hit = sphere->Intersect(ray, 1e-8f, closestHit.dist);

        if (hit.dist < closestHit.dist)
            closestHit = hit;
    }
    return closestHit;
}

__host__ __device__
glm::vec3 getRayColor(const Ray& ray, RenderParams& params, hiprandState* state)
{
    Ray curRay = ray;
    glm::vec3 rayColor{ 1.0f };
    for (int i = 0; i <= params.settings.bounceLimit; i++)
    {
        HitInfo hit = hitScene(curRay, params);
        if (hit.DidHit())
        {
            if (!hit.material->ScatterRay(curRay, hit, rayColor, curRay, state))
                break;
            curRay.origin += curRay.dir * 1e-4f;
        }
        else
        {
            float a = 0.5f * (curRay.dir.y + 1.0f);
            //const glm::vec3 lightDir = glm::normalize(glm::vec3(1.0f));
            //float a = glm::dot(curRay.dir, lightDir);
            //a = 0.5f * a + 0.5f;
            //a *= a;
            auto c = (1.0f - a) * glm::vec3(1.0f) + a * glm::vec3(0.5f, 0.7f, 1.0f);
            return rayColor * c;
        }
    }

    return glm::vec3(0.0f);
}